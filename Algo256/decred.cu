#include "hip/hip_runtime.h"
/**
 * Blake-256 Decred 180-Bytes input Cuda Kernel (Tested on SM 5/5.2)
 *
 * Tanguy Pruvot - Feb 2016
 * Alexis Provos - Mar 2016
 */

#include <stdint.h>
#include <memory.h>
#include <miner.h>

extern "C" {
#include <sph/sph_blake.h>
}

/* threads per block */
#define TPB 640

/* hash by cpu with blake 256 */
extern "C" void decred_hash(void *output, const void *input){
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}

#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(16) c_data[32];
__constant__ uint32_t _ALIGN(16) c_h[ 2];
__constant__ uint32_t _ALIGN(16) c_xors[215];

/* 8 adapters max */
static uint32_t		*d_resNonce[MAX_GPUS];
static uint32_t		*h_resNonce[MAX_GPUS];

/* macro bodies */
#define pxorGS(a,b,c,d) { \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}

#define pxorGS2(a,b,c,d,a1,b1,c1,d1) {\
	v[ a]+= c_xors[i++] + v[ b];			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032);	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[ c]+= v[ d];					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);		v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); 		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxory1GS2(a,b,c,d,a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); 		v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); 		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}
#define pxory0GS2(a,b,c,d,a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); 		v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= (c_xors[i++]^nonce) + v[ b]; 		v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}
#define pxorx1GS2(a,b,c,d,a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); 		v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); 		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}
#define pxorx0GS2(a,b,c,d,a1,b1,c1,d1) { \
	v[ a]+= (c_xors[i++]^nonce) + v[ b]; 		v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); 		v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b]; 			v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); 	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[ c]+= v[ d]; 					v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); 		v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

__global__ __launch_bounds__(TPB,1)
void decred_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce,uint32_t *resNonce,const uint32_t highTarget){

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (thread < threads){

		uint32_t v[16];
		#pragma unroll 
		for(int i=0;i<16;i+=4){
			*(uint4*)&v[i] = *(uint4*)&c_data[ i];
		}

		const uint32_t nonce = startNonce + thread;
		int i=0;
		v[ 1]+= (nonce ^ 0x13198A2E);
		v[13] = __byte_perm(v[13] ^ v[1], 0, 0x0321);
		v[ 9]+= v[13];
		v[ 5] = ROTR32(v[5] ^ v[9], 7);
	
		v[ 1]+= c_xors[i++];// + v[ 6];
		v[ 0]+= v[5];
		v[12] = __byte_perm(v[12] ^ v[ 1], 0, 0x1032);		v[15] = __byte_perm(v[15] ^ v[ 0], 0, 0x1032);
		v[11]+= v[12];						v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 12);			v[ 5] = ROTR32(v[5] ^ v[10], 12);
		v[ 1]+= c_xors[i++] + v[ 6];				v[ 0]+= c_xors[i++] + v[ 5];
		v[12] = __byte_perm(v[12] ^ v[ 1], 0, 0x0321);		v[15] = __byte_perm(v[15] ^ v[ 0], 0, 0x0321);
		v[11]+= v[12];						v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 7);			v[ 5] = ROTR32(v[ 5] ^ v[10], 7);

		pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxory1GS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorx1GS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		pxorx1GS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorx1GS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxory1GS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxory1GS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorx1GS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//{  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxory0GS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorx0GS2(	2, 7, 8,13, 3, 4, 9,14);
		//{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },		
		pxory1GS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxory1GS2(	2, 7, 8,13, 3, 4, 9,14);
		//  { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		pxorGS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorx1GS2(	0, 5,10,15, 1, 6,11,12);pxorGS2(	2, 7, 8,13, 3, 4, 9,14);
		//{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 }
		pxorx1GS2(	0, 4, 8,12, 1, 5, 9,13);pxorGS2(	2, 6,10,14, 3, 7,11,15);pxorGS2(	0, 5,10,15, 1, 6,11,12);pxorGS(		2, 7, 8,13);

		if ((c_h[1]^v[15]) == v[7]){
			v[3] += c_xors[i++] + v[4];
			v[14] = __byte_perm(v[14] ^ v[3], 0, 0x1032);
			v[9] += v[14];
			v[4] = ROTR32(v[4] ^ v[9], 12);
			v[3] += c_xors[i++] + v[4];
			v[14] = __byte_perm(v[14] ^ v[3], 0, 0x0321);
			if(cuda_swab32((c_h[0]^v[6]^v[14]))<=highTarget){
				atomicMin(&resNonce[0], nonce);
				return;
			}
		}
	}
}

__host__
void decred_cpu_setBlock_52(const uint32_t *input){
/*
	Precompute everything possible and pass it on constant memory
*/	
	const sph_u32 _ALIGN(64) z[16] = {
		SPH_C32(0x243F6A88), SPH_C32(0x85A308D3), SPH_C32(0x13198A2E), SPH_C32(0x03707344), SPH_C32(0xA4093822), SPH_C32(0x299F31D0), SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
		SPH_C32(0x452821E6), SPH_C32(0x38D01377), SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C), SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD), SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
	};
	
	int i=0;
	sph_u32 _ALIGN(64) preXOR[215];	
	sph_u32 _ALIGN(64)   data[16];
	sph_u32 _ALIGN(64)      m[16];
	sph_u32 _ALIGN(64)      h[ 2];
	
	sph_blake256_context ctx;
	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 128);

	data[ 0] = ctx.H[0];	data[ 1] = ctx.H[1];
	data[ 2] = ctx.H[2];	data[ 3] = ctx.H[3];
	data[ 4] = ctx.H[4];	data[ 5] = ctx.H[5];
	data[ 8] = ctx.H[6];	data[12] = swab32(input[35]);
	data[13] = ctx.H[7];
	// pre swab32
	m[ 0] = swab32(input[32]);	m[ 1] = swab32(input[33]);
	m[ 2] = swab32(input[34]);	m[ 3] = 0;
	m[ 4] = swab32(input[36]);	m[ 5] = swab32(input[37]);
	m[ 6] = swab32(input[38]);	m[ 7] = swab32(input[39]);
	m[ 8] = swab32(input[40]);	m[ 9] = swab32(input[41]);
	m[10] = swab32(input[42]);	m[11] = swab32(input[43]);
	m[12] = swab32(input[44]);	m[13] = 0x80000001;
	m[14] = 0;			m[15] = 0x000005a0;
	
	h[ 0] = data[ 8];
	h[ 1] = data[13];
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_h),h, 8, 0, hipMemcpyHostToDevice));
	
	data[ 0]+= (m[ 0] ^ z[1]) + data[ 4];
	data[12]  = SPH_ROTR32(z[4] ^ SPH_C32(0x5A0) ^ data[ 0], 16);

	data[ 8] = z[0]+data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 12);
	data[ 0]+= (m[ 1] ^ z[0]) + data[ 4];
	data[12] = SPH_ROTR32(data[12] ^ data[ 0],8);
	data[ 8]+= data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 7);
	
	data[ 1]+= (m[ 2] ^ z[3]) + data[ 5];
	data[13] = SPH_ROTR32((z[5] ^ SPH_C32(0x5A0)) ^ data[ 1], 16);
	data[ 9] = z[1]+data[13];
	data[ 5] = SPH_ROTR32(data[ 5] ^ data[ 9], 12);
	data[ 1]+= data[ 5]; //+nonce ^ ...
	
	data[ 2]+= (m[ 4] ^ z[5]) + h[ 0];
	data[14] = SPH_ROTR32(z[6] ^ data[ 2],16);
	data[10] = z[2] + data[14];
	data[ 6] = SPH_ROTR32(h[ 0] ^ data[10], 12);
	data[ 2]+= (m[ 5] ^ z[4]) + data[ 6];	
	data[14] = SPH_ROTR32(data[14] ^ data[ 2], 8);
	data[10]+= data[14];
	data[ 6] = SPH_ROTR32(data[ 6] ^ data[10], 7);

	data[ 3]+= (m[ 6] ^ z[7]) + h[ 1];
	data[15] = SPH_ROTR32(z[7] ^ data[ 3],16);
	data[11] = z[3] + data[15];
	data[ 7] = SPH_ROTR32(h[ 1] ^ data[11], 12);
	data[ 3]+= (m[ 7] ^ z[6]) + data[ 7];
	data[15] = SPH_ROTR32(data[15] ^ data[ 3],8);
	data[11]+= data[15];
	data[ 7] = SPH_ROTR32(data[11] ^ data[ 7], 7);
	data[ 0]+= m[ 8] ^ z[9];
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, 64, 0, hipMemcpyHostToDevice));

#define precalcXORGS(x,y) { \
	preXOR[i++]= (m[x] ^ z[y]); \
	preXOR[i++]= (m[y] ^ z[x]); \
}
#define precalcXORGS2(x,y,x1,y1){\
	preXOR[i++] = (m[ x] ^ z[ y]);\
	preXOR[i++] = (m[x1] ^ z[y1]);\
	preXOR[i++] = (m[ y] ^ z[ x]);\
	preXOR[i++] = (m[y1] ^ z[x1]);\
}
	precalcXORGS(10,11);
	preXOR[ 0]+=data[ 6];
	preXOR[i++] = (m[9] ^ z[8]);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);

	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);

	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);
	precalcXORGS2( 4, 0,15, 8);

	precalcXORGS2( 9, 0, 5, 7);
	precalcXORGS2( 2, 4,10,15);
	precalcXORGS2(14, 1,11,12);
	precalcXORGS2( 6, 8, 3,13);
	precalcXORGS2( 2,12, 6,10);
	precalcXORGS2( 0,11, 8, 3);
	
	precalcXORGS2( 4,13, 7, 5);
	precalcXORGS2(15,14, 1, 9);
	precalcXORGS2(12, 5, 1,15);
	precalcXORGS2(14,13, 4,10);
	precalcXORGS2( 0, 7, 6, 3);
	precalcXORGS2( 9, 2, 8,11);
	precalcXORGS2(13,11, 7,14);
	precalcXORGS2(12, 1, 3, 9);
	precalcXORGS2( 5, 0,15, 4);
	precalcXORGS2( 8, 6, 2,10);
	precalcXORGS2( 6,15,14, 9);
	precalcXORGS2(11, 3, 0, 8);
	precalcXORGS2(12, 2,13, 7);
	precalcXORGS2( 1, 4,10, 5);
	precalcXORGS2(10, 2, 8, 4);
	precalcXORGS2( 7, 6, 1, 5);
	precalcXORGS2(15,11, 9,14);
	precalcXORGS2( 3,12,13, 0);
	precalcXORGS2( 0, 1, 2, 3);
	precalcXORGS2( 4, 5, 6, 7);
	precalcXORGS2( 8, 9,10,11);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);
	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);

	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);	
	precalcXORGS( 4, 0);
	precalcXORGS(15, 8);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_xors), preXOR, 215*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

/* ############################################################################################################################### */

static bool init[MAX_GPUS] = { 0 };

// nonce position is different in decred
#define DCR_NONCE_OFT32 35

extern "C" int scanhash_decred(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done){
	uint32_t _ALIGN(64) endiandata[48];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *pnonce = &pdata[DCR_NONCE_OFT32];

	const uint32_t first_nonce = *pnonce;
	const uint32_t targetHigh = (opt_benchmark?0x1ULL:ptarget[6]);
	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 29 : 25;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);
	
	const dim3 grid((throughput + TPB-1)/(TPB));
	const dim3 block(TPB);
	
	int rc = 0;

	if (opt_benchmark) {
		ptarget[6] = swab32(0xff);
	}
	if (!init[thr_id]){
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id],  sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}
	hipMemset(d_resNonce[thr_id], 0xff, sizeof(uint32_t));
	
	memcpy(endiandata, pdata, 180);

	decred_cpu_setBlock_52(endiandata);
	
	do {
		// GPU HASH
		decred_gpu_hash_nonce <<<grid, block>>> (throughput,(*pnonce), d_resNonce[thr_id],targetHigh);
		hipDeviceSynchronize();
		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
		if (h_resNonce[thr_id][0] != UINT32_MAX){
			rc = 1;
//			work_set_target_ratio(work, vhashcpu);
			*hashes_done = (*pnonce) - first_nonce + throughput;
			work->nonces[0] = swab32(h_resNonce[thr_id][0]);
			*pnonce = work->nonces[0];
			return 1;
		}
		*pnonce += throughput;

	} while (!work_restart[thr_id].restart && (uint64_t)max_nonce > (uint64_t)throughput + (uint64_t)(*pnonce));

	*hashes_done = (*pnonce) - first_nonce;
	MyStreamSynchronize(NULL, 0, device_map[thr_id]);
	return rc;
}

// cleanup
extern "C" void free_decred(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();
	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
